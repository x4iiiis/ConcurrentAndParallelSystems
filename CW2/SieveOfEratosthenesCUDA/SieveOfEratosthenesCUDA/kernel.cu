#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <chrono>
#include <vector>

using namespace std;
using namespace chrono;

void cuda_info()
{
	//Get CUDA device
	int device;
	hipGetDevice(&device);

	//Get CUDA device properties
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	//Display properties
	cout << "Name: " << properties.name << endl;
	cout << "CUDA Capability: " << properties.major << "." << properties.minor << endl;
	cout << "Cores: " << properties.multiProcessorCount << endl;
	cout << "Memory: " << properties.totalGlobalMem / (1024 * 1024) << "MB" << endl;
	cout << "Clock Freq: " << properties.clockRate / 1000 << "MHz" << endl;
}


__global__ void Eratosthenes(bool *A)// , bool *B)
{
	//Get block index
	unsigned int block_idx = blockIdx.x;
	//Get thread index
	unsigned int thread_idx = threadIdx.x;
	//Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	//Get the thread's unique ID - (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;


	int upperBound = 1000000000;
	int upperBoundSquareRoot = (int)sqrt((double)upperBound);
	for (int m = 2; m <= upperBoundSquareRoot; m++)
	{
		if (!A[m])
		{
			cout << m << " ";			//Comment these out as they slow down performance
			//results << m << endl;		//R//Outputting the prime numbers to the text file

			for (int k = m * m; k <= upperBound; k += m)
			{
				A[k] = true;
			}
		}
	}
	for (int m = upperBoundSquareRoot; m <= upperBound; m++)
	{
		if (!A[m])
		{
			cout << m << " ";			//Comment these out as they slow down performance
			//results << m << endl;		//R//Outputting the prime numbers to the text file
		}
	}


	
	
	
	
	//Add corresponding locations of A and B and store in C
	//B[idx] = A[idx];	//Think this is how we "return" the result
}

int main()
{
	//Initialise CUDA - select device
	hipSetDevice(0);

	//8.2
	cuda_info();


	//SieveOfEratosthenes
	const unsigned int ELEMENTS = 1000000000;
	//Change upperBound to ELEMENTS at some point
	const unsigned int upperBound = 1000000000;

	int upperBoundSquareRoot = (int)sqrt((double)upperBound);
	bool *isComposite = new bool[upperBound + 1];
	memset(isComposite, 0, sizeof(bool) * (upperBound + 1));









	//Create host memory
	auto data_size = sizeof(bool) * ELEMENTS;
	vector<bool> A(ELEMENTS);	//Input array
	//vector<bool> B(ELEMENTS);	//Output array

	//Initialise input data
	for (unsigned int i = 0; i < ELEMENTS; i++)
	{
		A[i] = false; 
	}

	//Declare buffers
	bool *buffer_A;//, *buffer_B;

	//Initialise Buffers
	hipMalloc((void**)&buffer_A, data_size);
	//hipMalloc((void**)&buffer_B, data_size);

	//Write host data to device
	hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);

	//Run kernel with one thread for each element
	//First value is number of blocks, second is threads per block.Max 1024 threads per block
	Eratosthenes<<<ELEMENTS / 1024, 1024>>>(buffer_A);// , buffer_B);
	//Wait for kernel to complete
	hipDeviceSynchronize();
	//Read output buffer back to the host
	//hipMemcpy(&B[0], buffer_B, data_size, hipMemcpyDeviceToHost);



	//Clean up resources
	hipFree(buffer_A);
	//hipFree(buffer_B);
	
	return 0;
}
#include "hip/hip_runtime.h"
// 8.1.cpp : Defines the entry point for the console application.
//

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>


using namespace std;

//8.2
void cuda_info()
{
	//Get CUDA device
	int device;
	hipGetDevice(&device);

	//Get CUDA device properties
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	//Display properties
	cout << "Name: " << properties.name << endl;
	cout << "CUDA Capability: " << properties.major << "." << properties.minor << endl;
	cout << "Cores: " << properties.multiProcessorCount << endl;
	cout << "Memory: " << properties.totalGlobalMem / (1024 * 1024) << "MB" << endl;
	cout << "Clock Freq: " << properties.clockRate / 1000 << "MHz" << endl;
}

//8.3
__global__ void vecadd(const int *A, const int *B, int *C)
{
	//Get block index
	unsigned int block_idx = blockIdx.x;
	//Get thread index
	unsigned int thread_idx = threadIdx.x;
	//Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	//Get the thread's unique ID - (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	//Add corresponding locations of A and B and store in C
	C[idx] = A[idx] + B[idx];
}

int main()
{
	//Initialise CUDA - select device
	hipSetDevice(0);

	//8.2
	cuda_info();


	//From OpenCL
	const unsigned int ELEMENTS = 2048;

	//8.4
	//Create host memory
	auto data_size = sizeof(int) * ELEMENTS;
	vector<int> A(ELEMENTS);	//Input array
	vector<int> B(ELEMENTS);	//Input array
	vector<int> C(ELEMENTS);	//Output array
	//Initialise input data
	for (unsigned int i = 0; i < ELEMENTS; i++)
	{
		A[i] = B[i] = i;
	}
	//Declare buffers
	int *buffer_A, *buffer_B, *buffer_C;
	//Initialise buffers
	hipMalloc((void**)&buffer_A, data_size);
	hipMalloc((void**)&buffer_B, data_size);
	hipMalloc((void**)&buffer_C, data_size);
	//Write host data to device
	hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_B, &B[0], data_size, hipMemcpyHostToDevice);

	
	//8.5
	//Run kernel with one thread for each element
	//First value is number of blocks, second is threads per block.Max 1024 threads per block
	vecadd<<<ELEMENTS / 1024, 1024 >>>(buffer_A, buffer_B, buffer_C);
	//Wait for kernel to complete
	hipDeviceSynchronize();
	//Read output buffer back to the host
	hipMemcpy(&C[0], buffer_C, data_size, hipMemcpyDeviceToHost);

	//From OpenCL
	//Verify the output
	auto result = true;
	int i = 0;
	//Iterate through each value in result array
	for (auto &e : C)
	{
		//Check value
		if (e != i + i)
		{
			result = false;
			break;
		}
		i++;
	}
	//Check if result is true and display accordingly
	if (result)
	{
		cout << "Output is correct" << endl;
	}
	else
	{
		cout << "Output is incorrect" << endl;
	}

	//Clean up resources
	hipFree(buffer_A);
	hipFree(buffer_B);
	hipFree(buffer_C);


	return 0;
}

